#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "common.cuh"

__global__ void assignColorKernel(int64_t *color, hiprandStateMtgp32 *state, uint8_t *done,
                                  ptrdiff_t nNodes) {
  KERNEL_LOOP(i, nNodes) {
    if (color[i] < 0) {
      color[i] = (hiprand_uniform(&state[0]) < 0.53406) - 2;  // blue = -1, red = -2
      *done = 0;
    }
  }
}

int THCGreedy_assignColor(THCState *state, THCudaLongTensor *color) {
  int64_t *colorData = THCudaLongTensor_data(state, color);
  ptrdiff_t nNodes = THCudaLongTensor_nElement(state, color);
  uint8_t* d_done; hipMalloc(&d_done, sizeof(uint8_t)); hipMemset(d_done, 1, sizeof(uint8_t));
  KERNEL_RUN(assignColorKernel, nNodes, colorData, THCRandom_generatorStates(state), d_done);
  uint8_t done; hipMemcpy(&done, d_done, sizeof(uint8_t), hipMemcpyDeviceToHost);
  hipFree(d_done);
  return done;
}
