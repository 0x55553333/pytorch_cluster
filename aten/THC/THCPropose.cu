#include "hip/hip_runtime.h"
#include "common.cuh"

__global__ void proposeKernel(int64_t *color, int64_t *prop, int64_t *row, int64_t *col,
                              int64_t *deg, int64_t *cumDeg, ptrdiff_t nNodes) {
  KERNEL_LOOP(i, nNodes) {
    if (color[i] != -1) continue; // Only visit blue nodes.
    ptrdiff_t c;
    for (ptrdiff_t e = cumDeg[i] - deg[i]; e < cumDeg[i]; e++) {
      c = col[e];
      if (color[c] == -2) {  // Red neighbor found.
        prop[i] = c;  // Propose!
        break;
      }
    }
    if (prop[i] < 0) color[i] = i;  // Mark node as dead.
  }
}

void THCGreedy_propose(THCState *state, THCudaLongTensor *color, THCudaLongTensor *prop,
                       THCudaLongTensor *row, THCudaLongTensor *col, THCudaLongTensor *deg,
                       THCudaLongTensor *cumDeg) {
  ptrdiff_t nNodes = THCudaLongTensor_nElement(state, color);
  int64_t *colorData = THCudaLongTensor_data(state, color);
  int64_t *propData = THCudaLongTensor_data(state, prop);
  int64_t *rowData = THCudaLongTensor_data(state, row);
  int64_t *colData = THCudaLongTensor_data(state, col);
  int64_t *degData = THCudaLongTensor_data(state, deg);
  int64_t *cumDegData = THCudaLongTensor_data(state, cumDeg);
  KERNEL_RUN(proposeKernel, nNodes, colorData, propData, rowData, colData, degData, cumDegData);
}
