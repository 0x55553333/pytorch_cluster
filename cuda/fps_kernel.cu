#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include "atomics.cuh"
#include "utils.cuh"

#define THREADS 1024

template <typename scalar_t>
__global__ void
fps_kernel(scalar_t *__restrict__ x, int64_t *__restrict__ cum_deg,
           int64_t *__restrict__ cum_k, int64_t *__restrict__ start,
           scalar_t *__restrict__ dist, scalar_t *__restrict__ tmp_dist,
           int64_t *__restrict__ out, size_t dim) {

  const size_t batch_idx = blockIdx.x;
  const size_t idx = threadIdx.x;
  const size_t stride = blockDim.x; // == THREADS

  const size_t start_idx = cum_deg[batch_idx];
  const size_t end_idx = cum_deg[batch_idx + 1];

  int64_t old = start_idx + start[batch_idx];

  if (idx == 0) {
    out[cum_k[batch_idx]] = old;
  }

  for (ptrdiff_t m = cum_k[batch_idx] + 1; m < cum_k[batch_idx + 1]; m++) {

    for (ptrdiff_t n = start_idx + idx; n < end_idx; n += stride) {
      tmp_dist[n] = 0;
    }

    __syncthreads();
    for (ptrdiff_t i = start_idx * dim + idx; i < end_idx * dim; i += stride) {
      scalar_t d = x[(old * dim) + (i % dim)] - x[i];
      atomicAdd(&tmp_dist[i / dim], d * d);
    }

    __syncthreads();
    for (ptrdiff_t n = start_idx + idx; n < end_idx; n += stride) {
      dist[n] = min(dist[n], tmp_dist[n]);
    }
  }
}

at::Tensor fps_cuda(at::Tensor x, at::Tensor batch, float ratio, bool random) {
  auto batch_sizes = (int64_t *)malloc(sizeof(int64_t));
  hipMemcpy(batch_sizes, batch[-1].data<int64_t>(), sizeof(int64_t),
             hipMemcpyDeviceToHost);
  auto batch_size = batch_sizes[0] + 1;

  auto deg = degree(batch, batch_size);
  auto cum_deg = at::cat({at::zeros(1, deg.options()), deg.cumsum(0)}, 0);
  auto k = (deg.toType(at::kFloat) * ratio).round().toType(at::kLong);
  auto cum_k = at::cat({at::zeros(1, k.options()), k.cumsum(0)}, 0);

  at::Tensor start;
  if (random) {
    start = at::rand(batch_size, x.options());
    start = (start * deg.toType(at::kFloat)).toType(at::kLong);
  } else {
    start = at::zeros(batch_size, k.options());
  }

  auto dist = at::full(x.size(0), 1e38, x.options());
  auto tmp_dist = at::empty(x.size(0), x.options());

  auto k_sum = (int64_t *)malloc(sizeof(int64_t));
  hipMemcpy(k_sum, cum_k[-1].data<int64_t>(), sizeof(int64_t),
             hipMemcpyDeviceToHost);
  auto out = at::empty(k_sum[0], k.options());

  AT_DISPATCH_FLOATING_TYPES(x.type(), "fps_kernel", [&] {
    fps_kernel<scalar_t><<<batch_size, THREADS>>>(
        x.data<scalar_t>(), cum_deg.data<int64_t>(), cum_k.data<int64_t>(),
        start.data<int64_t>(), dist.data<scalar_t>(), tmp_dist.data<scalar_t>(),
        out.data<int64_t>(), x.size(1));
  });

  return dist;
}

// at::Tensor ifp_cuda(at::Tensor x, at::Tensor batch, float ratio) {
//   AT_DISPATCH_FLOATING_TYPES(x.type(), "ifp_kernel", [&] {
//     ifp_kernel<scalar_t><<<BLOCKS(x.numel()), THREADS>>>(
//         x.data<scalar_t>(), batch.data<int64_t>(), ratio, x.numel());
//   });

//   return x;
// }

// __global__ void ifps_kernel() {}

// // x: [N, F]
// // count: [B]
// // batch: [N]
// // tmp min distances: [N]
// // start node idx

// // we parallelize over n times f
// // parallelization over n times f: We can compute distances over atomicAdd
// // each block corresponds to a batch

// __global__ void farthestpointsamplingKernel(int b, int n, int m,
//                                             const float *__restrict__
//                                             dataset, float *__restrict__
//                                             temp, int *__restrict__ idxs) {
//   // dataset: [N*3] entries
//   // b: batch-size
//   // n: number of nodes
//   // m: number of sample points

//   if (m <= 0)
//     return;
//   const int BlockSize = 512;
//   __shared__ float dists[BlockSize];
//   __shared__ int dists_i[BlockSize];
//   const int BufferSize = 3072;
//   __shared__ float buf[BufferSize * 3];
//   for (int i = blockIdx.x; i < b; i += gridDim.x) { // iterate over all
//   batches?
//     int old = 0;
//     if (threadIdx.x == 0)
//       idxs[i * m + 0] = old;
//     for (int j = threadIdx.x; j < n; j += blockDim.x) { // iterate over all n
//       temp[blockIdx.x * n + j] = 1e38;
//     }
//     for (int j = threadIdx.x; j < min(BufferSize, n) * 3; j += blockDim.x) {
//       buf[j] = dataset[i * n * 3 + j];
//     }
//     __syncthreads();
//     for (int j = 1; j < m; j++) {
//       int besti = 0;
//       float best = -1;
//       float x1 = dataset[i * n * 3 + old * 3 + 0];
//       float y1 = dataset[i * n * 3 + old * 3 + 1];
//       float z1 = dataset[i * n * 3 + old * 3 + 2];
//       for (int k = threadIdx.x; k < n; k += blockDim.x) {
//         float td = temp[blockIdx.x * n + k];
//         float x2, y2, z2;
//         if (k < BufferSize) {
//           x2 = buf[k * 3 + 0];
//           y2 = buf[k * 3 + 1];
//           z2 = buf[k * 3 + 2];
//         } else {
//           x2 = dataset[i * n * 3 + k * 3 + 0];
//           y2 = dataset[i * n * 3 + k * 3 + 1];
//           z2 = dataset[i * n * 3 + k * 3 + 2];
//         }
//         float d = (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) +
//                   (z2 - z1) * (z2 - z1);
//         float d2 = min(d, td);
//         if (d2 != td)
//           temp[blockIdx.x * n + k] = d2;
//         if (d2 > best) {
//           best = d2;
//           besti = k;
//         }
//       }
//       dists[threadIdx.x] = best;
//       dists_i[threadIdx.x] = besti;
//       for (int u = 0; (1 << u) < blockDim.x; u++) {
//         __syncthreads();
//         if (threadIdx.x < (blockDim.x >> (u + 1))) {
//           int i1 = (threadIdx.x * 2) << u;
//           int i2 = (threadIdx.x * 2 + 1) << u;
//           if (dists[i1] < dists[i2]) {
//             dists[i1] = dists[i2];
//             dists_i[i1] = dists_i[i2];
//           }
//         }
//       }
//       __syncthreads();
//       old = dists_i[0];
//       if (threadIdx.x == 0)
//         idxs[i * m + j] = old;
//     }
//   }
// }
